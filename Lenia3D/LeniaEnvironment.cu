#include "hip/hip_runtime.h"
#include "LeniaEnvironment.cuh"
#include "KernelFunctions.cuh"

#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <fstream>

namespace {
	constexpr int kThreadsPerBlock = 8;
}

LeniaEnvironment::LeniaEnvironment(std::string file, int fieldSize, int kernelRadius, int stepsPerTimeUnit, float mu, float sigma,
	KernelType kernelType, GrowthType growthType) :
	fieldSize(fieldSize), kernelSize(2 * kernelRadius + 1), padSize(kernelRadius),
	paddedFieldSize(fieldSize + 2 * padSize),
	volume(fieldSize * fieldSize * fieldSize),
	volumePadded(paddedFieldSize * paddedFieldSize * paddedFieldSize),
	volumeKernel(kernelSize * kernelSize * kernelSize),
	mu(mu), sigma(sigma), 
	stepsPerTimeUnit(stepsPerTimeUnit), timeDelta(1.0 / stepsPerTimeUnit) {

	// malloc arrays
	imgCpu = (float*)calloc(volume, sizeof(*imgCpu));
	res = hipMalloc((void**)(&imgGpu), volumePadded * sizeof(*imgGpu));
	assert(res == hipSuccess);
	res = hipMalloc((void**)(&growth), volume * sizeof(*growth));
	assert(res == hipSuccess);
	res = hipMalloc((void**)(&kernel), volumeKernel * sizeof(*kernel));
	assert(res == hipSuccess);

	// set kernel launch params
	int gridSize = ceil(float(fieldSize) / float(kThreadsPerBlock));
	gridDim = dim3(gridSize, gridSize, gridSize);
	blockDim = dim3(kThreadsPerBlock, kThreadsPerBlock, kThreadsPerBlock);

	// initialize field
	std::ifstream input(file, std::ios::binary);
	input.read((char *)imgCpu, volume * sizeof(*imgCpu));

	// move field to gpu
	hipMemcpy3DParms cpuToGpu{ 0 };
	cpuToGpu.srcPtr = make_hipPitchedPtr(imgCpu,
		sizeof(*imgCpu) * fieldSize, fieldSize, fieldSize);
	cpuToGpu.dstPtr = make_hipPitchedPtr(imgGpu,
		sizeof(*imgGpu) * paddedFieldSize, paddedFieldSize, paddedFieldSize);
	cpuToGpu.extent = make_hipExtent(fieldSize * sizeof(*imgCpu), fieldSize, 
		fieldSize);
	cpuToGpu.srcPos = make_hipPos(0 * sizeof(*imgCpu), 0, 0);
	cpuToGpu.dstPos = make_hipPos(padSize * sizeof(*imgGpu), padSize, padSize);
	cpuToGpu.kind = hipMemcpyHostToDevice;
	res = hipMemcpy3D(&cpuToGpu);
	assert(res == hipSuccess);

	// initialize convolution kernel

	// power of 2 nearest to volumeKernel and smaller than it
	int sumReductionBufSize = volumeKernel;
	sumReductionBufSize--;
	sumReductionBufSize |= sumReductionBufSize >> 1;
	sumReductionBufSize |= sumReductionBufSize >> 2;
	sumReductionBufSize |= sumReductionBufSize >> 4;
	sumReductionBufSize |= sumReductionBufSize >> 8;
	sumReductionBufSize |= sumReductionBufSize >> 16;
	sumReductionBufSize++;
	sumReductionBufSize >>= 1;

	// malloc array for sum reduction
	float* sumReductionBuf;
	res = hipMalloc((void**)(&sumReductionBuf), sumReductionBufSize * sizeof(*sumReductionBuf));
	assert(res == hipSuccess);

	// calculate kernel
	switch (kernelType) {
	case KernelType::kUniform:
		setUniformKernel <<< gridDim, blockDim >>> (kernel, kernelSize, sumReductionBuf, sumReductionBufSize);
		break;
	case KernelType::kMove:
		setMoveKernel <<< gridDim, blockDim >>> (kernel, kernelSize, sumReductionBuf, sumReductionBufSize);
		break;
	case KernelType::kExponential:
		setExponentialKernel <<< gridDim, blockDim >>> (kernel, kernelSize, sumReductionBuf, sumReductionBufSize);
		break;
	case KernelType::kPolynomial:
		setPolynomialKernel <<< gridDim, blockDim >>> (kernel, kernelSize, sumReductionBuf, sumReductionBufSize);
		break;
	case KernelType::kRectangular:
		setRectangularKernel <<< gridDim, blockDim >>> (kernel, kernelSize, sumReductionBuf, sumReductionBufSize);
		break;
	case KernelType::kGol:
		setGolKernel <<< gridDim, blockDim >>> (kernel, kernelSize, sumReductionBuf, sumReductionBufSize);
		break;
	}

	// free array for sum reduction
	res = hipFree(sumReductionBuf);
	assert(res == hipSuccess);

	// set growth type function
	switch (growthType) {
	case GrowthType::kDouble:
		applyGrowth = applyDoubleGrowth;
		break;
	case GrowthType::kExponential:
		applyGrowth = applyExponentialGrowth;
		break;
	case GrowthType::kPolynomial:
		applyGrowth = applyPolynomialGrowth;
		break;
	case GrowthType::kRectangular:
		applyGrowth = applyRectangularGrowth;
		break;
	}
	res = hipGetLastError();
	assert(res == hipSuccess);

	// initialize memcpy params for padding
	params = MemcpyParams(fieldSize, paddedFieldSize, padSize, imgGpu);
};

void LeniaEnvironment::Convolute() {
	convolute <<< gridDim, blockDim >>> (
		imgGpu, kernel, growth, paddedFieldSize, kernelSize, fieldSize);
}

void LeniaEnvironment::SetPadding() {
	res = hipMemcpy3D(&(params.rowLo));
	assert(res == hipSuccess);
	res = hipMemcpy3D(&(params.rowHi));
	assert(res == hipSuccess);
	res = hipMemcpy3D(&(params.colLo));
	assert(res == hipSuccess);
	res = hipMemcpy3D(&(params.colHi));
	assert(res == hipSuccess);
	res = hipMemcpy3D(&(params.layLo));
	assert(res == hipSuccess);
	res = hipMemcpy3D(&(params.layHi));
	assert(res == hipSuccess);
}

void LeniaEnvironment::ApplyGrowth() {
	(*applyGrowth) <<< gridDim, blockDim >>> (
		growth, fieldSize, mu, sigma);
}

void LeniaEnvironment::SumAndCut() {
	sumAndCut <<< gridDim, blockDim >>> (
		imgGpu, growth, fieldSize, padSize, paddedFieldSize, timeDelta);
}

void LeniaEnvironment::Step() {
	SetPadding();
	Convolute();
	ApplyGrowth();
	SumAndCut();
}

float* LeniaEnvironment::GetResult() {
	hipMemcpy3DParms gpuToCpu{ 0 };
	gpuToCpu.dstPtr = make_hipPitchedPtr(imgCpu,
		sizeof(*imgCpu) * fieldSize, fieldSize, fieldSize);
	gpuToCpu.srcPtr = make_hipPitchedPtr(imgGpu,
		sizeof(*imgGpu) * paddedFieldSize, paddedFieldSize, paddedFieldSize);
	gpuToCpu.extent = make_hipExtent(fieldSize * sizeof(*imgCpu), fieldSize,
		fieldSize);
	gpuToCpu.dstPos = make_hipPos(0 * sizeof(*imgCpu), 0, 0);
	gpuToCpu.srcPos = make_hipPos(padSize * sizeof(*imgCpu), padSize, padSize);
	gpuToCpu.kind = hipMemcpyDeviceToHost;
	res = hipMemcpy3D(&gpuToCpu);
	assert(res == hipSuccess);

	return imgCpu;
}

float* LeniaEnvironment::StepTimeUnit() {
	for (int i = 0; i < stepsPerTimeUnit; i++) {
		Step();
	}
	return GetResult();
}

void LeniaEnvironment::PutStepTimeUnit(std::string file) {
	float* result = StepTimeUnit();
	std::ofstream output(file, std::ios::binary);
	output.write((char *)result, volume * sizeof(*result));
}

void LeniaEnvironment::PutResult(std::string file) {
	float* result = GetResult();
	std::ofstream output(file, std::ios::binary);
	output.write((char*)result, volume * sizeof(*result));
}


LeniaEnvironment::~LeniaEnvironment() {
	free(imgCpu);
	res = hipFree(imgGpu);
	assert(res == hipSuccess);
	res = hipFree(growth);
	assert(res == hipSuccess);
	res = hipFree(kernel);
	assert(res == hipSuccess);
};

LeniaEnvironment::MemcpyParams::MemcpyParams(int fieldSize, int paddedFieldSize, int padSize, float* ptr) :
	rowLo{ 0 }, rowHi{ 0 }, colLo{ 0 }, colHi{ 0 }, layLo{ 0 }, layHi{ 0 } {

	hipPitchedPtr pitchedPtr = make_hipPitchedPtr(ptr,
		sizeof(float) * paddedFieldSize, paddedFieldSize, paddedFieldSize);
	rowLo.srcPtr = pitchedPtr;
	rowHi.srcPtr = pitchedPtr;
	colLo.srcPtr = pitchedPtr;
	colHi.srcPtr = pitchedPtr;
	layLo.srcPtr = pitchedPtr;
	layHi.srcPtr = pitchedPtr;
	rowLo.dstPtr = pitchedPtr;
	rowHi.dstPtr = pitchedPtr;
	colLo.dstPtr = pitchedPtr;
	colHi.dstPtr = pitchedPtr;
	layLo.dstPtr = pitchedPtr;
	layHi.dstPtr = pitchedPtr;

	hipExtent rowExtent = make_hipExtent(padSize * sizeof(float), fieldSize,
		fieldSize);
	hipExtent colExtent = make_hipExtent(paddedFieldSize * sizeof(float), padSize,
		fieldSize);
	hipExtent layExtent = make_hipExtent(paddedFieldSize * sizeof(float), paddedFieldSize,
		padSize);
	rowLo.extent = rowExtent;
	rowHi.extent = rowExtent;
	colLo.extent = colExtent;
	colHi.extent = colExtent;
	layLo.extent = layExtent;
	layHi.extent = layExtent;

	rowLo.srcPos = make_hipPos((paddedFieldSize - 2 * padSize) * sizeof(float), padSize, padSize);
	rowLo.dstPos = make_hipPos(0 * sizeof(float), padSize, padSize);
	rowHi.srcPos = make_hipPos(padSize * sizeof(float), padSize, padSize);
	rowHi.dstPos = make_hipPos((paddedFieldSize - padSize) * sizeof(float), padSize, padSize);
	colLo.srcPos = make_hipPos(0 * sizeof(float), paddedFieldSize - 2 * padSize, padSize);
	colLo.dstPos = make_hipPos(0 * sizeof(float), 0, padSize);
	colHi.srcPos = make_hipPos(0 * sizeof(float), padSize, padSize);
	colHi.dstPos = make_hipPos(0 * sizeof(float), paddedFieldSize - padSize, padSize);
	layLo.srcPos = make_hipPos(0 * sizeof(float), 0, paddedFieldSize - 2 * padSize);
	layLo.dstPos = make_hipPos(0 * sizeof(float), 0, 0);
	layHi.srcPos = make_hipPos(0 * sizeof(float), 0, padSize);
	layHi.dstPos = make_hipPos(0 * sizeof(float), 0, paddedFieldSize - padSize);

	rowLo.kind = hipMemcpyDeviceToDevice;
	rowHi.kind = hipMemcpyDeviceToDevice;
	colLo.kind = hipMemcpyDeviceToDevice;
	colHi.kind = hipMemcpyDeviceToDevice;
	layLo.kind = hipMemcpyDeviceToDevice;
	layHi.kind = hipMemcpyDeviceToDevice;
};
