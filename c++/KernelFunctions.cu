#include "hip/hip_runtime.h"
#include "KernelFunctions.cuh"

#include <iostream>

#include <hip/hip_cooperative_groups.h>

__global__ void convolute(float* img, float* kernel, float* growth,
	int paddedFieldSize, int kernelSize, int fieldSize)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int lay = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < fieldSize && col < fieldSize && lay < fieldSize) {
		float sum = 0.0;

		int paddedFieldSizeSqr = paddedFieldSize * paddedFieldSize;
		int fieldSizeSqr = fieldSize * fieldSize;
		int kernelSizeSqr = kernelSize * kernelSize;

		for (int kernelLay = 0; kernelLay < kernelSize; kernelLay++) {
			for (int kernelCol = 0; kernelCol < kernelSize; kernelCol++) {
				for (int kernelRow = 0; kernelRow < kernelSize; kernelRow++) {
					int imgRow = row + kernelRow;
					int imgCol = col + kernelCol;
					int imgLay = lay + kernelLay;

					//printf("res[%d][%d][%d]: img[%d][%d][%d](%f) * ker[%d][%d][%d](%f)\n", lay, col, row,
					//	imgPad, imgCol, imgRow,
					//	image[imgPad * imageRowsCols + imgCol * imageRows + imgRow],
					//	kernelLay, kernelCol, kernelRow,
					//	kernel[kernelLay * kernelRowsCols + kernelCol * kernelRows + kernelRow]);
					sum +=
						img[imgLay * paddedFieldSizeSqr + imgCol * paddedFieldSize + imgRow] *
						kernel[kernelLay * kernelSizeSqr + kernelCol * kernelSize + kernelRow];
				}
			}
		}
		// printf("growth[%d][%d][%d] = %f\n",
		// 	lay, col, row, sum);
		growth[lay * fieldSizeSqr +
			col * fieldSize +
			row] = sum;
	}
}

__global__ void sumAndCut(float* img, float* growth,
	int fieldSize, int padSize, int paddedFieldSize, float timeDelta) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int lay = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < fieldSize && col < fieldSize && lay < fieldSize) {
		int paddedFieldSizeSqr = paddedFieldSize * paddedFieldSize;
		int fieldSizeSqr = fieldSize * fieldSize;

		int imgRow = row + padSize;
		int imgCol = col + padSize;
		int imgLay = lay + padSize;

		int imgIdx = imgLay * paddedFieldSizeSqr + imgCol * paddedFieldSize + imgRow;

		float sum = img[imgIdx]
			+ timeDelta * growth[lay * fieldSizeSqr + col * fieldSize + row];
		sum = sum > 1.0 ? 1.0 : (sum < 0.0 ? 0.0 : sum);
		img[imgIdx] = sum;
		// printf("img[%d][%d][%d] (aka img[%d]) += growth[%d][%d][%d](%f)\n", 
		// 	imgLay, imgCol, imgRow, imgIdx, lay, col, row, growth[lay * fieldSizeSqr + col * fieldSize + row]);
		// img[imgIdx] = growth[lay * fieldSizeSqr + col * fieldSize + row];
	}
}

__global__ void applyDoubleGrowth(float* growth, int fieldSize, float mu, float sigma) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int lay = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < fieldSize && col < fieldSize && lay < fieldSize) {
		int fieldSizeSqr = fieldSize * fieldSize;
		int idx = lay * fieldSizeSqr + col * fieldSize + row;

		float orig = growth[idx];
		orig *= 2;
		growth[idx] = orig;
	}
}

__global__ void applyExponentialGrowth(float* growth, int fieldSize, float mu, float sigma) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int lay = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < fieldSize && col < fieldSize && lay < fieldSize) {
		int fieldSizeSqr = fieldSize * fieldSize;
		int idx = lay * fieldSizeSqr + col * fieldSize + row;

		float orig = growth[idx];
		orig = 2.0 * expf( - (orig - mu) * (orig - mu) / (2.0 * sigma * sigma)) - 1.0;
		growth[idx] = orig;
	}
}

__global__ void applyPolynomialGrowth(float* growth, int fieldSize, float mu, float sigma) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int lay = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < fieldSize && col < fieldSize && lay < fieldSize) {
		int fieldSizeSqr = fieldSize * fieldSize;
		int idx = lay * fieldSizeSqr + col * fieldSize + row;

		float orig = growth[idx];
		if (orig >= mu - 3.0 * sigma && orig <= mu + 3.0 * sigma) {
			orig = 2.0 * powf(1.0 - (orig - mu) * (orig - mu) / (9.0 * sigma * sigma), 4.0) - 1.0;
		}
		else {
			orig = -1.0;
		}
		growth[idx] = orig;
	}
}

__global__ void applyRectangularGrowth(float* growth, int fieldSize, float mu, float sigma) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int lay = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < fieldSize && col < fieldSize && lay < fieldSize) {
		int fieldSizeSqr = fieldSize * fieldSize;
		int idx = lay * fieldSizeSqr + col * fieldSize + row;

		float orig = growth[idx];
		if (orig >= mu - sigma && orig <= mu + sigma) {
			orig = 1.0;
		}
		else {
			orig = -1.0;
		}
		growth[idx] = orig;
	}
}
