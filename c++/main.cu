#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>						// hipDeviceSynchronize()
#include <iostream>
#include <iomanip>
#include <chrono>

#include "LeniaEnvironment.cuh"
#include "EnvironmentCpu.cuh"

void printResult(float* result, int fieldSize) {
	for (size_t lay = 0; lay < fieldSize; lay++) {
		for (size_t col = 0; col < fieldSize; col++)
		{
			for (size_t row = 0; row < fieldSize; row++)
			{
				std::cout << std::fixed << std::setw(3) << std::setprecision(1)
					<< result[lay * fieldSize * fieldSize + col * fieldSize + row] << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main(int argc, char ** argv) {
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	int fieldSize = std::stoi(argv[1]);
	int kernelRadius = std::stoi(argv[2]);
	int stepsPerTimeUnit = std::stoi(argv[3]);
	int timeUnits = std::stoi(argv[4]);
	float mu = std::stof(argv[5]);
	float sigma = std::stof(argv[6]);

	if (argv[10][0] == 'g') {
		LeniaEnvironment::KernelType kernelType;
		switch (argv[7][0]) {
		case 'u':
			kernelType = LeniaEnvironment::KernelType::kUniform;
			break;
		case 'm':
			kernelType = LeniaEnvironment::KernelType::kMove;
			break;
		case 'e':
			kernelType = LeniaEnvironment::KernelType::kExponential;
			break;
		case 'p':
			kernelType = LeniaEnvironment::KernelType::kPolynomial;
			break;
		case 'r':
			kernelType = LeniaEnvironment::KernelType::kRectangular;
			break;
		case 'g':
			kernelType = LeniaEnvironment::KernelType::kGol;
			break;
		default:
			std::cerr << "Unknown kernel type: " << argv[7][0] << std::endl;
			throw std::runtime_error("Unknown kernel type");
		}

		LeniaEnvironment::GrowthType growthType;
		switch (argv[8][0]) {
		case 'd':
			growthType = LeniaEnvironment::GrowthType::kDouble;
			break;
		case 'e':
			growthType = LeniaEnvironment::GrowthType::kExponential;
			break;
		case 'p':
			growthType = LeniaEnvironment::GrowthType::kPolynomial;
			break;
		case 'r':
			growthType = LeniaEnvironment::GrowthType::kRectangular;
			break;
		default:
			std::cerr << "Unknown growth type" << std::endl;
			throw std::runtime_error("Unknown growth type");
		}

		LeniaEnvironment env(argv[9], fieldSize, kernelRadius, stepsPerTimeUnit, mu, sigma,
			kernelType, growthType);

		int num_width = int(ceilf(log10f(timeUnits + 1)));
		std::string file = "res/" + std::string(num_width, '0') + ".bin";
		env.PutResult(file);

		for (int i = 0; i < timeUnits; i++) {
			std::string idx = std::to_string(i + 1);
			file = "res/" + std::string(num_width - idx.length(), '0') + idx + ".bin";
			env.PutStepTimeUnit(file);
		}
	}
	if (argv[10][0] == 'c') {
		EnvironmentCpu::KernelType kernelType;
		switch (argv[7][0]) {
		case 'u':
			kernelType = EnvironmentCpu::KernelType::kUniform;
			break;
		case 'm':
			kernelType = EnvironmentCpu::KernelType::kMove;
			break;
		case 'e':
			kernelType = EnvironmentCpu::KernelType::kExponential;
			break;
		case 'p':
			kernelType = EnvironmentCpu::KernelType::kPolynomial;
			break;
		case 'r':
			kernelType = EnvironmentCpu::KernelType::kRectangular;
			break;
		case 'g':
			kernelType = EnvironmentCpu::KernelType::kGol;
			break;
		default:
			std::cerr << "Unknown kernel type: " << argv[7][0] << std::endl;
			throw std::runtime_error("Unknown kernel type");
		}

		EnvironmentCpu::GrowthType growthType;
		switch (argv[8][0]) {
		case 'd':
			growthType = EnvironmentCpu::GrowthType::kDouble;
			break;
		case 'e':
			growthType = EnvironmentCpu::GrowthType::kExponential;
			break;
		case 'p':
			growthType = EnvironmentCpu::GrowthType::kPolynomial;
			break;
		case 'r':
			growthType = EnvironmentCpu::GrowthType::kRectangular;
			break;
		default:
			std::cerr << "Unknown growth type" << std::endl;
			throw std::runtime_error("Unknown growth type");
		}

		EnvironmentCpu env(argv[9], fieldSize, kernelRadius, stepsPerTimeUnit, mu, sigma,
			kernelType, growthType);

		int num_width = int(ceilf(log10f(timeUnits + 1)));
		std::string file = "res/" + std::string(num_width, '0') + ".bin";
		env.PutResult(file);

		for (int i = 0; i < timeUnits; i++) {
			std::string idx = std::to_string(i + 1);
			file = "res/" + std::string(num_width - idx.length(), '0') + idx + ".bin";
			env.PutStepTimeUnit(file);
		}
	}

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() / 1000.0 << " s" << std::endl;
}